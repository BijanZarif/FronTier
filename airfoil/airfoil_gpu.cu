#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "airfoil_gpu.cuh"

__global__ void preset_kernel_spring(SPRING_VERTEX*,
			double**,double*,double*,int*,int);
__global__ void set_kernel_spring(SPRING_VERTEX*,
			double*,double*,double*,double**,double**,
			double*,double*,double**,double**,
			double*,double*,double**,double**,
			double*,double**,int,int);
__global__ void pos_to_old(double**,double**,
			double**,double**,int,int);
__global__ void comp_spring_accel(SPRING_VERTEX*,double**,int,int);
__global__ void RK_1(double**,double**,
			double**,double**,double**,double**,
			double**,double,int,int);
__global__ void RK_2(double**,double**,
			double**,double**,double**,double**,
			double**,double,int,int);
__global__ void RK_3(double**,double**,
			double**,double**,double**,double**,
			double**,double,int,int);
__global__ void RK_4(double**,double**,
			double**,double**,double**,double**,
			double**,double,int,int);
__global__ void Update_x_new(SPRING_VERTEX*,double**,double,int,int);
__global__ void Set_impul(SPRING_VERTEX*,double*,int);
__device__ void dev_comp_spring_accel(SPRING_VERTEX*,double*,int);

extern void gpu_spring_solver(
	SPRING_VERTEX *sv,
	int dim,
	int size,
	int n_tan,
	double dt)
{
	static SPRING_VERTEX *dev_sv;
	static double *dev_x_store,*dev_v_store;
	static double *dev_f_store,*dev_impul_store;
	static double *dev_x_old_store,*dev_v_old_store;
	static double *dev_x_new_store,*dev_v_new_store;
	static double *dev_accel_store;
	static double *dev_k,*dev_len0;
	static int *dev_ix;
	static double **dev_x_pos,**dev_v_pos;
	static double **dev_x_old,**dev_v_old;
	static double **dev_x_new,**dev_v_new;
	static double **dev_accel;
	static double **dev_x_nb_store;
	static int first = 1;
	int i,j,n;
	int total_num_nb;
	double *data;
	int *ix;
	int TPB = 256;
	int NB = (size+TPB-1)/TPB;

	static double *x_data;

	(void) printf("Entering gpu_spring_solver()\n");
	if (first)
	{
	    first = 0;
	    hipMalloc((void**)&dev_sv,size*sizeof(SPRING_VERTEX));
	    hipMalloc((void**)&dev_x_store,dim*size*sizeof(double));
	    hipMalloc((void**)&dev_v_store,dim*size*sizeof(double));
	    hipMalloc((void**)&dev_f_store,dim*size*sizeof(double));
	    hipMalloc((void**)&dev_impul_store,3*size*sizeof(double));
	    hipMalloc((void**)&dev_x_pos,size*sizeof(double*));
	    hipMalloc((void**)&dev_v_pos,size*sizeof(double*));
	    /* The folllowing are for internally used data */
	    hipMalloc((void**)&dev_x_old_store,dim*size*sizeof(double));
	    hipMalloc((void**)&dev_v_old_store,dim*size*sizeof(double));
	    hipMalloc((void**)&dev_x_old,size*sizeof(double*));
	    hipMalloc((void**)&dev_v_old,size*sizeof(double*));
	    hipMalloc((void**)&dev_x_new_store,dim*size*sizeof(double));
	    hipMalloc((void**)&dev_v_new_store,dim*size*sizeof(double));
	    hipMalloc((void**)&dev_x_new,size*sizeof(double*));
	    hipMalloc((void**)&dev_v_new,size*sizeof(double*));
	    hipMalloc((void**)&dev_accel_store,dim*size*sizeof(double));
	    hipMalloc((void**)&dev_accel,size*sizeof(double*));

	    total_num_nb = 0;
	    for (i = 0; i < size; i++)
	    	total_num_nb += sv[i].num_nb;

	    data = (double*)malloc(total_num_nb*sizeof(double));
	    ix = (int*)malloc(total_num_nb*sizeof(int));
	    x_data = (double*) malloc(dim*size*sizeof(double));

	    /* This call will copy all direct elements of sv */
	    /* including num_nb,m,lambda. */
	    hipMemcpy(dev_sv,sv,size*sizeof(SPRING_VERTEX),
				hipMemcpyHostToDevice);

	    n = 0;
	    for (i = 0; i < size; i++)
	    for (j = 0; j < sv[i].num_nb; j++)
	    {
	    	data[n++] = sv[i].k[j];
	    }
	    hipMalloc((void**)&dev_k,total_num_nb*sizeof(double));
	    hipMemcpy(dev_k,data,total_num_nb*sizeof(double),
				hipMemcpyHostToDevice);
	    n = 0;
	    for (i = 0; i < size; i++)
	    for (j = 0; j < sv[i].num_nb; j++)
	    {
	    	data[n++] = sv[i].len0[j];
	    }
	    hipMalloc((void**)&dev_len0,total_num_nb*sizeof(double));
	    hipMemcpy(dev_len0,data,total_num_nb*sizeof(double),
				hipMemcpyHostToDevice);
	    n = 0;
	    for (i = 0; i < size; i++)
	    for (j = 0; j < sv[i].num_nb; j++)
	    {
	    	ix[n++] = sv[i].ix_nb[j];
	    }
	    hipMalloc((void**)&dev_ix,total_num_nb*sizeof(int));
	    hipMemcpy(dev_ix,ix,total_num_nb*sizeof(int),
				hipMemcpyHostToDevice);
	    hipMalloc((void**)&dev_x_nb_store,total_num_nb*sizeof(double*));
	    /* The following call will set x,v,x_nb,k,len0 for sv */
	    
	    preset_kernel_spring<<<1,1>>>(dev_sv,
			dev_x_nb_store,dev_k,dev_len0,dev_ix,size);

	    set_kernel_spring<<<NB,TPB>>>(dev_sv,
			dev_x_store,dev_v_store,dev_f_store,
                        dev_x_pos,dev_v_pos,
                        dev_x_old_store,dev_v_old_store,
			dev_x_old,dev_v_old,
			dev_x_new_store,dev_v_new_store,
			dev_x_new,dev_v_new, 
			dev_accel_store,dev_accel,
			size,dim);

	}

	/* The following two copy calls input data for each time step */

        for (i = 0; i < size; i++)
        for (j = 0; j < dim; j++)
        {
            x_data[i*dim+j] = sv[i].x[j];
        }
	hipMemcpy(dev_x_store,x_data,dim*size*sizeof(double),
				hipMemcpyHostToDevice);
        for (i = 0; i < size; i++)
        for (j = 0; j < dim; j++)
        {
            x_data[i*dim+j] = sv[i].v[j];
        }
	hipMemcpy(dev_v_store,x_data,dim*size*sizeof(double),
				hipMemcpyHostToDevice);
	pos_to_old<<<NB,TPB>>>(dev_x_old,dev_x_pos,
				dev_v_old,dev_v_pos,size,dim);
	comp_spring_accel<<<NB,TPB>>>(dev_sv,dev_accel,size,dim);

	for (n = 0; n < n_tan; ++n)
        {
	    RK_1<<<NB,TPB>>>(
			dev_x_new,dev_v_new,dev_x_pos,dev_v_pos,
			dev_x_old,dev_v_old,dev_accel,dt,size,dim);
	    comp_spring_accel<<<NB,TPB>>>(dev_sv,dev_accel,size,dim);

	    RK_2<<<NB,TPB>>>(
			dev_x_new,dev_v_new,dev_x_pos,dev_v_pos,
			dev_x_old,dev_v_old,dev_accel,dt,size,dim);
	    comp_spring_accel<<<NB,TPB>>>(dev_sv,dev_accel,size,dim);

	    RK_3<<<NB,TPB>>>(
			dev_x_new,dev_v_new,dev_x_pos,dev_v_pos,
			dev_x_old,dev_v_old,dev_accel,dt,size,dim);
	    comp_spring_accel<<<NB,TPB>>>(dev_sv,dev_accel,size,dim);

	    RK_4<<<NB,TPB>>>(
			dev_x_new,dev_v_new,dev_x_pos,dev_v_pos,
			dev_x_old,dev_v_old,dev_accel,dt,size,dim);
	    Update_x_new<<<NB,TPB>>>(dev_sv,dev_x_new,dt,size,dim);
	    pos_to_old<<<NB,TPB>>>(dev_x_pos,dev_x_new,
			dev_v_pos,dev_v_new,size,dim);
	    if (n != n_tan-1)
            {
		comp_spring_accel<<<NB,TPB>>>(dev_sv,dev_accel,size,dim);
		
		pos_to_old<<<NB,TPB>>>(dev_x_old,
			dev_x_pos,dev_v_old,dev_v_pos,size,dim);
            } 
	}
	Set_impul<<<NB,TPB>>>(dev_sv,dev_impul_store,size);
	hipMemcpy(x_data,dev_impul_store,dim*size*sizeof(double),
				hipMemcpyDeviceToHost);
        for (i = 0; i < size; i++)
        for (j = 0; j < 3; j++)
        {
            sv[i].ext_impul[j] = x_data[i*3+j];
        }
	hipMemcpy(x_data,dev_f_store,dim*size*sizeof(double),
				hipMemcpyDeviceToHost);
        for (i = 0; i < size; i++)
        for (j = 0; j < dim; j++)
        {
            sv[i].f[j] = x_data[i*dim+j];
        }
	hipMemcpy(x_data,dev_x_store,dim*size*sizeof(double),
				hipMemcpyDeviceToHost);
        for (i = 0; i < size; i++)
        for (j = 0; j < dim; j++)
        {
            sv[i].x[j] = x_data[i*dim+j];
        }
	hipMemcpy(x_data,dev_v_store,dim*size*sizeof(double),
				hipMemcpyDeviceToHost);
        for (i = 0; i < size; i++)
        for (j = 0; j < dim; j++)
        {
            sv[i].v[j] = x_data[i*dim+j];
        }
	(void) printf("Leaving gpu_spring_solver()\n");
}	/* end gpu_spring_solver */

__global__ void preset_kernel_spring(
	SPRING_VERTEX *sv,
	double **dev_x_nb_store,
	double *dev_k,
	double *dev_len0,
	int *dev_ix,
	int size)
{
	int i,n = 0;

	for (i = 0; i < size; ++i)
	{
	    sv[i].k = dev_k + n; 
	    sv[i].len0 = dev_len0 + n; 
	    sv[i].ix_nb = dev_ix + n; 
	    sv[i].x_nb = dev_x_nb_store + n; 
	    n += sv[i].num_nb;
	}
}	/* end preset_kernel_spring */
	
__global__ void set_kernel_spring(
	SPRING_VERTEX *sv,
	double *dev_x_store,
	double *dev_v_store,
	double *dev_f_store,
	double **dev_x_pos,
	double **dev_v_pos,
	double *dev_x_old_store,
	double *dev_v_old_store,
	double **dev_x_old,
	double **dev_v_old,
	double *dev_x_new_store,
	double *dev_v_new_store,
	double **dev_x_new,
	double **dev_v_new,
	double *dev_accel_store,
	double **dev_accel,
	int size,
	int dim)
{
	int i,j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < size)
	{
	    sv[i].x = dev_x_store + i*dim; 
	    sv[i].v = dev_v_store + i*dim; 
	    sv[i].f = dev_f_store + i*dim; 
	    dev_x_pos[i] = dev_x_store + i*dim; 
	    dev_v_pos[i] = dev_v_store + i*dim; 
	    dev_x_old[i] = dev_x_old_store + i*dim; 
	    dev_v_old[i] = dev_v_old_store + i*dim; 
	    dev_x_new[i] = dev_x_new_store + i*dim; 
	    dev_v_new[i] = dev_v_new_store + i*dim; 
	    dev_accel[i] = dev_accel_store + i*dim; 
	    for (j = 0; j < sv[i].num_nb; ++j)
	    {
	    	sv[i].x_nb[j] = dev_x_store + sv[i].ix_nb[j]*dim; 
	    }
	}
}	/* end set_kernel_spring */
	
__global__ void pos_to_old(
	double **dev_x_old,
	double **dev_x_pos,
	double **dev_v_old,
	double **dev_v_pos,
	int size,
	int dim)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < size)
        {
            for ( int j = 0; j < dim; ++j)
            {
        	dev_x_old[i][j] = dev_x_pos[i][j];
        	dev_v_old[i][j] = dev_v_pos[i][j];
            }
	}
}	/* end pos_to_old */

__global__ void comp_spring_accel(
	SPRING_VERTEX *sv,
	double **dev_accel,
	int size,
	int dim)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
        {
	    dev_comp_spring_accel(&(sv[i]),dev_accel[i],dim);
	}
}

__device__ void dev_comp_spring_accel(
	SPRING_VERTEX *sv,
	double *accel,
	int dim)
{
	int i,k;
	double len,vec[3];
	for (k = 0; k < dim; ++k) 
            accel[k] = 0.0;
        for (i = 0; i < sv->num_nb; ++i)
        {
            len = 0.0;
            for (k = 0; k < dim; ++k)
            {
                vec[k] = sv->x_nb[i][k] - sv->x[k];
                len += vec[k]*vec[k];
            }
            len = sqrt(len);
            for (k = 0; k < dim; ++k)
            {
                vec[k] /= len;
                accel[k] += sv->k[i]*((len - sv->len0[i])*vec[k])/sv->m;
            }
        }
        for (k = 0; k < dim; ++k)
        {
            sv->f[k] += accel[k]*sv->m;
        }
        for (k = 0; k < dim; ++k)
        {
            accel[k] += -sv->lambda*sv->v[k]/sv->m;
        }
}	/* end dev_compute_spring_accel */

__global__ void	RK_1(
	double **dev_x_new,
	double **dev_v_new,
	double **dev_x_pos,
	double **dev_v_pos,
	double **dev_x_old,
	double **dev_v_old,
	double **dev_accel,
	double dt,
	int size,
	int dim)
{
	int i,j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
        {
	    for (j = 0; j < dim; ++j)
            {
                dev_x_new[i][j] = dev_x_old[i][j] + dt*dev_v_old[i][j]/6.0;
                dev_v_new[i][j] = dev_v_old[i][j] + dt*dev_accel[i][j]/6.0;
                dev_x_pos[i][j] = dev_x_old[i][j] + 0.5*dev_v_old[i][j]*dt;
                dev_v_pos[i][j] = dev_v_old[i][j] + 0.5*dev_accel[i][j]*dt;
            }
	}
}

__global__ void	RK_2(
	double **dev_x_new,
	double **dev_v_new,
	double **dev_x_pos,
	double **dev_v_pos,
	double **dev_x_old,
	double **dev_v_old,
	double **dev_accel,
	double dt,
	int size,
	int dim)
{
	int i,j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
        {
	    for (j = 0; j < dim; ++j)
            {
		dev_x_new[i][j] += dt*dev_v_pos[i][j]/3.0;
		dev_v_new[i][j] += dt*dev_accel[i][j]/3.0;
		dev_x_pos[i][j] = dev_x_old[i][j] + 0.5*dev_v_pos[i][j]*dt;
		dev_v_pos[i][j] = dev_v_old[i][j] + 0.5*dev_accel[i][j]*dt;
            }
	}
}

__global__ void	RK_3(
	double **dev_x_new,
	double **dev_v_new,
	double **dev_x_pos,
	double **dev_v_pos,
	double **dev_x_old,
	double **dev_v_old,
	double **dev_accel,
	double dt,
	int size,
	int dim)
{
	int i,j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
        {
	    for (j = 0; j < dim; ++j)
            {
		dev_x_new[i][j] += dt*dev_v_pos[i][j]/3.0;
		dev_v_new[i][j] += dt*dev_accel[i][j]/3.0;
		dev_x_pos[i][j] = dev_x_old[i][j] + dev_v_pos[i][j]*dt;
		dev_v_pos[i][j] = dev_v_old[i][j] + dev_accel[i][j]*dt;
            }
	}
}

__global__ void	RK_4(
	double **dev_x_new,
	double **dev_v_new,
	double **dev_x_pos,
	double **dev_v_pos,
	double **dev_x_old,
	double **dev_v_old,
	double **dev_accel,
	double dt,
	int size,
	int dim)
{
	int i,j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
        {
	    for (j = 0; j < dim; ++j)
            {
		dev_x_new[i][j] += dt*dev_v_pos[i][j]/6.0;
		dev_v_new[i][j] += dt*dev_accel[i][j]/6.0;
            }
	}
}

__global__ void Update_x_new(
	SPRING_VERTEX *sv,
	double **dev_x_new,
	double dt,
	int size,
	int dim)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j;
	if (i < size)
        {
	    for (j = 0; j < dim; ++j)
	    {
		dev_x_new[i][j] += (sv[i].ext_impul[j]
			+ 0.5*sv[i].ext_accel[j]*dt)*dt;
		sv[i].ext_impul[j] += sv[i].ext_accel[j]*dt;
	    } 
	}
}	/* end Update_x_new */

__global__ void Set_impul(
	SPRING_VERTEX *sv,
	double *dev_impul_store,
	int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j;
        if (i < size)
        {
            for (j = 0; j < 3; ++j)
            {
		dev_impul_store[3*i+j] = sv[i].ext_impul[j];
	    }
	}
}	/* end Set_impul */
